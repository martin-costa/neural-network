#include "hip/hip_runtime.h"
#include "networkalgorithms.cuh"

//// __ Stochastic Gradient Descent using CPU __ ////

// sgd algorithm for training the neural network on the CPU
void stochasticDescentCPU(Network* net, int epochs, int miniBatchSize, double learnRate, DataSet<NumberData>* trainingData, DataSet<NumberData>* testData) {

  // run each epoch of training
  for (int i = 0; i < epochs; i++) {
    int batchCount = trainingData->getSize() / miniBatchSize;
    DataSet<NumberData>* miniBatches = trainingData->randomise().split(miniBatchSize);

    // update mini batches
    for (int j = 0; j < batchCount; j++) {
      updateMiniBatch(net, &miniBatches[j], learnRate);
    }

    delete[] miniBatches;

    // test the current weights and biases against the test data
    if (testData != nullptr) {
      std::cout << net->evaluate(*testData) << "/" << testData->getSize() << " numbers correctly classified\n";
    }
    std::cout << "Epoch " << i << " complete\n";
  }
}

// applies sgd using backprop to a mini batch
void updateMiniBatch(Network* net, DataSet<NumberData>* miniBatch, double learnRate) {

  Matrix* gradw = new Matrix[net->getLayerCount() - 1]();
  Vector* gradb = new Vector[net->getLayerCount() - 1]();

  Matrix* dgradw = new Matrix[net->getLayerCount() - 1]();
  Vector* dgradb = new Vector[net->getLayerCount() - 1]();

  for (int j = 0; j < net->getLayerCount() - 1; j++) {
    gradw[j] = Matrix(net->getWeights()[j].rows(), net->getWeights()[j].cols());
    gradb[j] = Vector(net->getBiases()[j].size());
  }

  // for each piece of training data
  for (int i = 0; i < miniBatch->getSize(); i++) {

    for (int j = 0; j < net->getLayerCount() - 1; j++) {
      dgradw[j] = Matrix(net->getWeights()[j].rows(), net->getWeights()[j].cols());
      dgradb[j] = Vector(net->getBiases()[j].size());
    }

    backpropagation(net, dgradw, dgradb, &(*miniBatch)[i].getData(), &(*miniBatch)[i].getResult());

    for (int j = 0; j < net->getLayerCount() - 1; j++) {
      gradw[j] += dgradw[j];
      gradb[j] += dgradb[j];
    }
  }

  for (int j = 0; j < net->getLayerCount() - 1; j++) {
    net->getWeights()[j] += gradw[j] * (-learnRate / (double)miniBatch->getSize());
    net->getBiases()[j] += gradb[j] * (-learnRate / (double)miniBatch->getSize());
  }

  delete[] gradw;
  delete[] gradb;
  delete[] dgradw;
  delete[] dgradb;
}

// backpropegation algorithm for calculating the gradient of the cost function
void backpropagation(Network* net, Matrix* dgradw, Vector* dgradb, Vector* image, Vector* result) {
  int layerCount = net->getLayerCount();

  Vector* As = new Vector[layerCount](); //activations
  Vector* Zs = new Vector[layerCount - 1](); //pre sigmoid activations

  // get activations of layers
  As[0] = *image;

  for (int i = 0; i < layerCount - 1; i++) {
    Zs[i] = net->getWeights()[i] * As[i] + net->getBiases()[i];
    As[i + 1] = Sigmoid::f(Zs[i]);
  }

  // backwards pass
  Vector delta = CrossEntropyCost::finalError(Zs[layerCount - 2], As[layerCount - 1], *result);

  dgradb[layerCount - 2] = delta;
  dgradw[layerCount - 2] = Matrix::toRowMat(delta) * Matrix::toColMat(As[layerCount - 2]);

  for (int i = 2; i < layerCount; i++) {
    delta = (transpose(net->getWeights()[layerCount - i]) * delta).schur(Sigmoid::fPrime(Zs[layerCount - i - 1]));
    dgradb[layerCount - i - 1] = delta;
    dgradw[layerCount - i - 1] = Matrix::toRowMat(delta) * Matrix::toColMat(As[layerCount - i - 1]);
  }

  delete[] As;
  delete[] Zs;
}

/*

//// __ Stochastic Gradient Descent using GPU __ ////

// sgd algorithm for training the neural network on the GPU
void __host__ stochasticDescentGPU(Network* net, int epochs, int miniBatchSize, double learnRate, DataSet<NumberData>* trainingData, DataSet<NumberData>* testData) {

  // get the network data onto the GPU

  // parameters
  int layerCount = net->getLayerCount();
  int* layersDev = 0;
  double* weightsDev = 0;
  double* biasesDev = 0;

  //data
  double* trainingDataDev = 0;
  double* testDataDev = 0;

  int* trainingLabelsDev = 0;
  int* testLabelsDev = 0;

  int* trainingDataPermDev = 0;
  int* testDataPermDev = 0;

  // allocate space and values to layersDev
  hipMalloc((void**)&layersDev, sizeof(int) * layerCount);
  hipMemcpy(layersDev, net->getLayers(), sizeof(int) * layerCount, hipMemcpyHostToDevice);

  // allocate space and values to weightsDev
  int weightsSize = 0;
  for (int i = 0; i < layerCount - 1; i++) {
    weightsSize += net->getLayers()[i + 1] * net->getLayers()[i];
  }
  hipMalloc((void**)&weightsDev, sizeof(double) * weightsSize);

  int j = 0;
  for (int i = 0; i < layerCount - 1; i++) {
    hipMemcpy(&weightsDev[j], &net->getWeights()[i][0], sizeof(double) * net->getLayers()[i + 1] * net->getLayers()[i], hipMemcpyHostToDevice);
    j += net->getLayers()[i + 1] * net->getLayers()[i];
  }

  //allocate space and values to biasesDev
  int biasesSize = 0;
  for (int i = 0; i < layerCount - 1; i++) {
    biasesSize += net->getLayers()[i + 1];
  }
  hipMalloc((void**)&biasesDev, sizeof(double) * biasesSize);

  j = 0;
  for (int i = 0; i < layerCount - 1; i++) {
    hipMemcpy(&biasesDev[j], &net->getBiases()[i][0], sizeof(double) * net->getLayers()[i + 1], hipMemcpyHostToDevice);
    j += net->getLayers()[i + 1];
  }

  // allcoate space and values to data
  int trainingDataSize = trainingData->getSize();
  int testDataSize = testData->getSize();
  int dataSize = (*trainingData)[0].getData().size();

  hipMalloc((void**)&trainingDataDev, trainingDataSize * sizeof(double) * dataSize);
  hipMalloc((void**)&testDataDev, testDataSize * sizeof(double) * dataSize);

  hipMalloc((void**)&trainingLabelsDev, sizeof(int) * trainingDataSize);
  hipMalloc((void**)&testLabelsDev, sizeof(int) * testDataSize);

  int *trainingLabelsHost = new int[trainingDataSize];
  int* testLabelsHost = new int[trainingDataSize];

  for (int i = 0; i < trainingDataSize; i++) {
    hipMemcpy(&trainingDataDev[i * dataSize], &(*trainingData)[i].getData()[0], sizeof(double) * dataSize, hipMemcpyHostToDevice);
    trainingLabelsHost[i] = (*trainingData)[i].getNumber();
  }
  hipMemcpy(trainingLabelsDev, trainingLabelsHost, sizeof(int) * trainingDataSize, hipMemcpyHostToDevice);

  for (int i = 0; i < testDataSize; i++) {
    hipMemcpy(&testDataDev[i * dataSize], &(*testData)[i].getData()[0], sizeof(double) * dataSize, hipMemcpyHostToDevice);
    testLabelsHost[i] = (*testData)[i].getNumber();
  }
  hipMemcpy(testLabelsDev, testLabelsHost, sizeof(int) * testDataSize, hipMemcpyHostToDevice);

  //hipMalloc((void**)&trainingDataPermDev, sizeof(double) * trainingData->getSize());
  //hipMalloc((void**)&testDataPermDev, sizeof(double) * testData->getSize());

  //int* trainingDataPermHost = permutation(trainingData->getSize());
  //int* testDataPermHost = permutation(trainingData->getSize());

  //hipMemcpy(trainingDataPermDev, trainingDataPermHost, sizeof(double) * trainingData->getSize(), hipMemcpyHostToDevice);
  //hipMemcpy(testDataPermDev, testDataPermHost, sizeof(double) * testData->getSize(), hipMemcpyHostToDevice);

  //delete[] trainingDataPermHost;
  //delete[] testDataPermHost;

  // run each epoch of training
  for (int i = 0; i < epochs; i++) {
    int batchCount = trainingData->getSize() / miniBatchSize;
    DataSet<NumberData>* miniBatches = trainingData->randomise().split(miniBatchSize);

    // update mini batches
    for (int j = 0; j < batchCount; j++) {
      updateMiniBatch(net, &miniBatches[j], learnRate);
    }

    delete[] miniBatches;

    // test the current weights and biases against the test data
    if (testData != nullptr) {
      std::cout << net->evaluate(*testData) << "/" << testData->getSize() << " numbers correctly classified\n";
    }
    std::cout << "Epoch " << i << " complete\n";
  }

  std::cout << evaluate(testDataDev, testLabelsHost, testDataSize, dataSize, &net->getLayers()[0], layerCount, weightsDev, biasesDev) << "\n";

  // pass parameters back to the CPU
  int weightOffset = 0;
  int biasOffset = 0;

  for (int i = 0; i < layerCount - 1; i++) {
    hipMemcpy(&net->getWeights()[i][0], &weightsDev[weightOffset], sizeof(double) * net->getLayers()[i + 1] * net->getLayers()[i], hipMemcpyDeviceToHost);
    hipMemcpy(&net->getBiases()[i][0], &biasesDev[biasOffset], sizeof(double)* net->getLayers()[i + 1], hipMemcpyDeviceToHost);

    weightOffset += net->getLayers()[i + 1] * net->getLayers()[i];
    biasOffset += net->getLayers()[i + 1];
  }

  // free up the GPU
  hipFree(layersDev);
  hipFree(weightsDev);
  hipFree(biasesDev);

  hipFree(trainingDataDev);
  hipFree(testDataDev);

  hipFree(trainingDataPermDev);
  hipFree(testDataPermDev);

  delete[] trainingLabelsHost;
  delete[] testLabelsHost;
}

// kernels for network
double __host__ __device__ sigmoidFunction(double x) {
  return 1.0 / (1.0 + expf(-x));
}

double* __host__ __device__ sigmoidActivation(double* v, int size) {

  dim3 blockDim(std::min(size, 1024));
  dim3 gridDim(getBlock(blockDim.x, size));

  sigmoidActivationKernel <<<gridDim, blockDim>>> (v, size);

  return v;
}

void __global__ sigmoidActivationKernel(double* v, int size) {
  int i = threadIdx.x + blockDim.x * blockIdx.x;

  if (i < size)
    v[i] = sigmoidFunction(v[i]);
}

double* __host__ feedForward(double* x, int* layersHost, int layerCount, double* weightsDev, double* biasesDev) {
  int weightOffset = layersHost[1] * layersHost[0];
  int biasOffset = layersHost[1];

  // -> CAUSING MEMORY LEAKS <- //
  double* a = sigmoidActivation(vectorAddOn(vectorMatMult(&weightsDev[0], x, layersHost[1], layersHost[0]), &biasesDev[0], layersHost[1]), layersHost[1]);

  for (int i = 1; i < layerCount - 1; i++) {
    a = sigmoidActivation(vectorAddOn(vectorMatMult(&weightsDev[weightOffset], a, layersHost[i + 1], layersHost[i]), &biasesDev[biasOffset], layersHost[i + 1]), layersHost[i + 1]);

    weightOffset += layersHost[i + 1] * layersHost[i];
    biasOffset += layersHost[i + 1];
  }
  return a;
}

int __host__ evaluate(double* testData, int* testLabels, int testDataSize, int dataSize, int* layersHost, int layerCount, double* weightsDev, double* biasesDev) {
  int j = 0;

  for (int i = 0; i < testDataSize; i++) {
    if (maxIndex(feedForward(&testData[i * dataSize], layersHost, layerCount, weightsDev, biasesDev), layersHost[layerCount - 1]) == testLabels[i])
      j++;
  }

  return j;
}

*/