
#include <hip/hip_runtime.h>
/*

#include "linear-algebra-kernels.cuh"

double* __host__ vectorMatMult(double* M, double* v, int m, int n) {

  double* u = 0;
  cudaMalloc((void**)&u, sizeof(double) * m);

  dim3 blockDim(std::min(m, 1024));
  dim3 gridDim(getBlock(blockDim.x, m));

  vectorMatMultKernel <<<gridDim, blockDim>>> (u, M, v, m, n);

  return u;
}

void __global__ vectorMatMultKernel(double* u, double* M, double* v, int m, int n) {
  int i = threadIdx.x + blockDim.x * blockIdx.x;

  if (i < m) {
    double temp = 0;

    for (int j = 0; j < n; j++) {
      temp += M[i + j * m] * v[j];
    }
    u[i] = temp;
  }
}

double* __host__ __device__ vectorAddOn(double* v, double* u, int size) {

  dim3 blockDim(std::min(size, 1024));
  dim3 gridDim(getBlock(blockDim.x, size));

  vectorAddOnKernel <<<gridDim, blockDim>>> (v, u, size);

  return v;
}

void __global__ vectorAddOnKernel(double* v, double* u, int size) {
  int i = threadIdx.x + blockDim.x * blockIdx.x;

  if (i < size)
    v[i] += u[i];
}

int __host__ __device__ maxIndex(double* v, int size) {

  int* idxDev = 0;
  cudaMalloc((void**)&idxDev, sizeof(int));
  maxIndexKernel <<<1, 1>>> (v, idxDev, size);

  int idx[] = {0};
  *idx = 0;

  cudaMemcpy(idx, idxDev, sizeof(int), cudaMemcpyDeviceToHost);
  cudaFree(idxDev);

  return *idx;
}

void __global__ maxIndexKernel(double* v, int* idx, int size) {

  double max = v[0];
  *idx = 0;

  for (int i = 1; i < size; i++) {
    if (v[i] >= max) {
      max = v[i];
      *idx = i;
    }
  }
}

void __global__ schurProductKernel(double* w, double* v, double* u, int size) {
  int i = threadIdx.x + blockDim.x * blockIdx.x;

  if (i < size)
    w[i] = v[i] * u[i];
}

*/